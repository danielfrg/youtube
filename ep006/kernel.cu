#include <cfloat>
#include <chrono>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>

// Error-checking macro
#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

/**
 * Kernel 1: Compute absolute difference: C[i] = |A[i] - B[i]|
 */
__global__ void absDiffKernel(const float *A, const float *B, float *C,
                              size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    C[i] = fabsf(A[i] - B[i]);
  }
}

/**
 * Kernel 2: Parallel reduction to find maximum in C.
 * Writes one value (the max) per thread block into partialMax[blockIdx.x].
 */
__global__ void reduceMaxKernel(const float *input, float *partialMax,
                                size_t N) {
  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  // Load elements into shared memory (or a very small sentinel if out of range)
  if (i < N) {
    sdata[tid] = input[i];
  } else {
    sdata[tid] = -FLT_MAX;
  }
  __syncthreads();

  // Reduction in shared memory
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();
  }

  // Write block result
  if (tid == 0) {
    partialMax[blockIdx.x] = sdata[0];
  }
}

/**
 * Utility function: parallel max reduction on d_in of size N.
 * The result is returned in *h_out on the host.
 */
void reduceMax(const float *d_in, float *h_out, size_t N, int blockSize) {
  int gridSize = (int)((N + blockSize - 1) / blockSize);

  // Allocate partial results
  float *d_partialMax = nullptr;
  CUDA_CHECK(hipMalloc(&d_partialMax, gridSize * sizeof(float)));

  // Shared memory size
  const int SHARED_BYTES = blockSize * sizeof(float);

  const float *d_currentIn = d_in;
  size_t currentSize = N;

  while (true) {
    reduceMaxKernel<<<gridSize, blockSize, SHARED_BYTES>>>(
        d_currentIn, d_partialMax, currentSize);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    if (gridSize == 1) {
      // Final max in d_partialMax[0]
      CUDA_CHECK(hipMemcpy(h_out, d_partialMax, sizeof(float),
                            hipMemcpyDeviceToHost));
      break;
    }

    // Prepare next pass
    currentSize = gridSize;
    gridSize = (gridSize + blockSize - 1) / blockSize;
    d_currentIn = d_partialMax;
  }

  CUDA_CHECK(hipFree(d_partialMax));
}

float compute(float *d_A, float *d_B, size_t size) {
  // Allocate device memory for result
  float *d_C = nullptr;
  CUDA_CHECK(hipMalloc(&d_C, size * sizeof(float)));

  // Launch abs-diff kernel
  const int BLOCK_SIZE = 256;
  int gridSize = (int)((size + BLOCK_SIZE - 1) / BLOCK_SIZE);

  absDiffKernel<<<gridSize, BLOCK_SIZE>>>(d_A, d_B, d_C, size);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  // Reduce to find maximum
  float max_diff = 0.0f;
  reduceMax(d_C, &max_diff, size, BLOCK_SIZE);

  // Clean up
  CUDA_CHECK(hipFree(d_C));

  return max_diff;
}

int main() {
  // Data Gen
  size_t size = 1 * (1 << 28);
  std::vector<float> h_A(size);
  std::vector<float> h_B(size);

  std::printf("Each vector contains %zu elements\n", size);

  size_t allocated_bytes = size * sizeof(float);
  std::printf("Each vector uses %zu bytes (%.2f GB)\n", allocated_bytes,
              allocated_bytes / (1024.0 * 1024.0 * 1024.0));

  // Fill the vectors with random values
  std::printf("Generating numbers...\n");
  std::mt19937 gen(69);
  std::normal_distribution<float> normal_dist(0.5f, 0.15f);

  auto rand_start = std::chrono::high_resolution_clock::now();
  for (size_t i = 0; i < size; i++) {
    h_A[i] = normal_dist(gen);
    h_B[i] = normal_dist(gen);
  }
  auto rand_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> rand_elapsed = rand_end - rand_start;
  std::printf("Time: Data generation: %.6f s\n", rand_elapsed.count());

  // Allocate GPU memory
  std::printf("Moving data to GPU...\n");
  float *d_A = nullptr;
  float *d_B = nullptr;
  float *d_C = nullptr; // to store |A - B|
  CUDA_CHECK(hipMalloc(&d_A, size * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_B, size * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_C, size * sizeof(float)));

  // Copy data to device and compute
  auto start_copy = std::chrono::high_resolution_clock::now();
  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), size * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, h_B.data(), size * sizeof(float),
                        hipMemcpyHostToDevice));
  auto end_copy = std::chrono::high_resolution_clock::now();
  double time_copy =
      std::chrono::duration<double>(end_copy - start_copy).count();
  std::printf("Time: Host to Device copy: %.6f s\n", time_copy);

  // Launch abs-diff kernel
  std::printf("Calculating...\n");
  auto start = std::chrono::high_resolution_clock::now();

  float max_diff = compute(d_A, d_B, size);

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  std::printf("Time: Compute: %.6f s\n", elapsed.count());

  std::printf("Max difference: %.6f\n", max_diff);

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  return 0;
}
