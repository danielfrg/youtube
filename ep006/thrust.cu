#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdio>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <vector>

int main() {
  // Data gen
  size_t size = 5 * (1 << 28);
  std::vector<float> a(size);
  std::vector<float> b(size);

  std::printf("Each vector contains %zu elements\n", a.size());

  size_t allocated_bytes_a = a.capacity() * sizeof(float);
  std::printf("Each vector allocated %zu bytes (%.2f GB)\n", allocated_bytes_a,
              allocated_bytes_a / (1024.0 * 1024.0 * 1024.0));

  // Fill the vectors with random values
  std::printf("Generating numbers...\n");
  std::mt19937 gen(69);
  std::normal_distribution<float> normal_dist(0.5f, 0.15f);

  auto rand_start = std::chrono::high_resolution_clock::now();
  for (size_t i = 0; i < size; i++) {
    a[i] = normal_dist(gen);
    b[i] = normal_dist(gen);
  }
  auto rand_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> rand_elapsed = rand_end - rand_start;
  std::printf("Time: Data generation: %.6f s\n", rand_elapsed.count());

  // Transfer to GPU
  std::printf("Transferring data to GPU...\n");
  auto transfer_start = std::chrono::high_resolution_clock::now();

  thrust::device_vector<float> d_a = a;
  thrust::device_vector<float> d_b = b;
  thrust::device_vector<float> d_diff(size);

  auto transfer_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> transfer_elapsed = transfer_end - transfer_start;
  std::printf("Time: Data transfer to GPU: %.6f s\n", transfer_elapsed.count());

  // Compute on GPU
  std::printf("Calculating on GPU...\n");
  auto start = std::chrono::high_resolution_clock::now();

  // Compute absolute differences
  thrust::transform(
      thrust::cuda::par, d_a.begin(), d_a.end(), d_b.begin(), d_diff.begin(),
      [] __host__ __device__(float x, float y) { return std::abs(x - y); });

  float max_diff = thrust::reduce(thrust::cuda::par, d_diff.begin(),
                                  d_diff.end(), 0.0f, thrust::maximum<float>());

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  std::printf("Time: Compute (GPU): %.6f s\n", elapsed.count());

  std::printf("Max difference: %.6f\n", max_diff);

  return 0;
}
