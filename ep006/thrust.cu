#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdio>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <vector>

// GPU version of compute function
float compute(const thrust::device_vector<float> &a,
              const thrust::device_vector<float> &b) {

  // Create a temporary vector to store the absolute differences
  thrust::device_vector<float> diff(a.size());

  // Calculate absolute differences between corresponding elements
  thrust::transform(thrust::device, a.begin(), a.end(), b.begin(), diff.begin(),
                    [] __device__(float x, float y) { return fabsf(x - y); });

  // Find the maximum difference using reduction
  float max_diff = thrust::reduce(thrust::device, diff.begin(), diff.end(),
                                  0.0f, thrust::maximum<float>());

  return max_diff;
}

int main() {
  // Data gen
  size_t size = 1 * (1 << 28);
  std::vector<float> a(size);
  std::vector<float> b(size);

  std::printf("Each vector contains %zu elements\n", a.size());

  size_t allocated_bytes_a = a.capacity() * sizeof(float);
  std::printf("Each vector allocated %zu bytes (%.2f GB)\n", allocated_bytes_a,
              allocated_bytes_a / (1024.0 * 1024.0 * 1024.0));

  // Fill the vectors with random values
  std::printf("Generating numbers...\n");
  std::mt19937 gen(69);
  std::normal_distribution<float> normal_dist(0.5f, 0.15f);

  auto rand_start = std::chrono::high_resolution_clock::now();
  for (size_t i = 0; i < size; i++) {
    a[i] = normal_dist(gen);
    b[i] = normal_dist(gen);
  }
  auto rand_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> rand_elapsed = rand_end - rand_start;
  std::printf("Time: Data generation: %.6f s\n", rand_elapsed.count());

  // Transfer to GPU
  std::printf("Transferring data to GPU...\n");
  auto transfer_start = std::chrono::high_resolution_clock::now();

  thrust::device_vector<float> d_a = a;
  thrust::device_vector<float> d_b = b;
  thrust::device_vector<float> d_diff(size);

  auto transfer_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> transfer_elapsed =
      transfer_end - transfer_start;
  std::printf("Time: Data transfer to GPU: %.6f s\n", transfer_elapsed.count());

  // Compute on GPU
  std::printf("Calculating on GPU...\n");
  auto start = std::chrono::high_resolution_clock::now();

  float max_diff = compute(d_a, d_b);

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  std::printf("Time: Compute (GPU): %.6f s\n", elapsed.count());

  std::printf("Max difference: %.6f\n", max_diff);

  return 0;
}
